#include "hip/hip_runtime.h"
#if defined(_WIN64)
    #define WIN32_NO_STATUS
    #include <windows.h>
    #undef WIN32_NO_STATUS
#endif

#include <thread>
#include <cinttypes>
#include <iomanip>
#include <iostream>
#include <mutex>
#include <queue>
#include <chrono>
#include <fstream>
#include <vector>

#include "secure_rand.h"
#include "structures.h"

#include "cpu_curve_math.h"
#include "cpu_keccak.h"
#include "cpu_math.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>

#define OUTPUT_BUFFER_SIZE 10000

#define BLOCK_SIZE 256U
#define THREAD_WORK (1U << 8)

__constant__ CurvePoint thread_offsets[BLOCK_SIZE];
__constant__ CurvePoint addends[THREAD_WORK - 1];
__device__ uint64_t device_memory[2 + OUTPUT_BUFFER_SIZE * 3];

__device__ int count_zero_bytes(uint32_t x) {
    int n = 0;
    n += ((x & 0xFF) == 0);
    n += ((x & 0xFF00) == 0);
    n += ((x & 0xFF0000) == 0);
    n += ((x & 0xFF000000) == 0);
    return n;
}

__device__ int score_zero_bytes(Address a) {
    int n = 0;
    n += count_zero_bytes(a.a);
    n += count_zero_bytes(a.b);
    n += count_zero_bytes(a.c);
    n += count_zero_bytes(a.d);
    n += count_zero_bytes(a.e);
    return n;
}

__device__ int score_leading_zeros(Address a) {
    int n = __clz(a.a);
    if (n == 32) {
        n += __clz(a.b);

        if (n == 64) {
            n += __clz(a.c);

            if (n == 96) {
                n += __clz(a.d);

                if (n == 128) {
                    n += __clz(a.e);
                }
            }
        }
    }

    return n >> 3;
}

#ifdef __linux__
    #define atomicMax_ul(a, b) atomicMax((unsigned long long*)(a), (unsigned long long)(b))
    #define atomicAdd_ul(a, b) atomicAdd((unsigned long long*)(a), (unsigned long long)(b))
#else
    #define atomicMax_ul(a, b) atomicMax(a, b)
    #define atomicAdd_ul(a, b) atomicAdd(a, b)
#endif

__device__ void handle_output(int score_method, Address a, uint64_t key, bool inv) {
    int score = 0;
    if (score_method == 0) { score = score_leading_zeros(a); }
    else if (score_method == 1) { score = score_zero_bytes(a); }

    if (score >= device_memory[1]) {
        atomicMax_ul(&device_memory[1], score);
        if (score >= device_memory[1]) {
            uint32_t idx = atomicAdd_ul(&device_memory[0], 1);
            if (idx < OUTPUT_BUFFER_SIZE) {
                device_memory[2 + idx] = key;
                device_memory[OUTPUT_BUFFER_SIZE + 2 + idx] = score;
                device_memory[OUTPUT_BUFFER_SIZE * 2 + 2 + idx] = inv;
            }
        }
    }
}

__device__ void handle_output2(int score_method, Address a, uint64_t key) {
    int score = 0;
    if (score_method == 0) { score = score_leading_zeros(a); }
    else if (score_method == 1) { score = score_zero_bytes(a); }

    if (score >= device_memory[1]) {
        atomicMax_ul(&device_memory[1], score);
        if (score >= device_memory[1]) {
            uint32_t idx = atomicAdd_ul(&device_memory[0], 1);
            if (idx < OUTPUT_BUFFER_SIZE) {
                device_memory[2 + idx] = key;
                device_memory[OUTPUT_BUFFER_SIZE + 2 + idx] = score;
            }
        }
    }
}

#include "address.h"
#include "contract_address.h"
#include "contract_address2.h"

int global_max_score = 0;
std::mutex global_max_score_mutex;
uint32_t GRID_SIZE = 1U << 15;

struct Message {
    uint64_t time;

    int status;
    int device_index;
    hipError_t error;

    double speed;
    int results_count;
    _uint256* results;
    int* scores;
};

std::queue<Message> message_queue;
std::mutex message_queue_mutex;


#define gpu_assert(call) { \
    hipError_t e = call; \
    if (e != hipSuccess) { \
        message_queue_mutex.lock(); \
        message_queue.push(Message{milliseconds(), 1, device_index, e}); \
        message_queue_mutex.unlock(); \
        if (thread_offsets_host != 0) { hipHostFree(thread_offsets_host); } \
        if (device_memory_host != 0) { hipHostFree(device_memory_host); } \
        hipDeviceReset(); \
        return; \
    } \
}

uint64_t milliseconds() {
    return (std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch())).count();
}


void host_thread(int device, int device_index, int score_method, int mode, Address create2_factory, _uint256 bytecode) {
    uint64_t GRID_WORK = ((uint64_t)BLOCK_SIZE * (uint64_t)GRID_SIZE * (uint64_t)THREAD_WORK);

    CurvePoint* block_offsets = 0;
    CurvePoint* offsets = 0;
    CurvePoint* thread_offsets_host = 0;

    uint64_t* device_memory_host = 0;
    uint64_t* max_score_host;
    uint64_t* output_counter_host;
    uint64_t* output_buffer_host;
    uint64_t* output_buffer2_host;
    uint64_t* output_buffer3_host;

    gpu_assert(hipSetDevice(device));

    gpu_assert(hipHostAlloc(&device_memory_host, (2 + OUTPUT_BUFFER_SIZE * 3) * sizeof(uint64_t), hipHostMallocDefault))
    output_counter_host = device_memory_host;
    max_score_host = device_memory_host + 1;
    output_buffer_host = max_score_host + 1;
    output_buffer2_host = output_buffer_host + OUTPUT_BUFFER_SIZE;
    output_buffer3_host = output_buffer2_host + OUTPUT_BUFFER_SIZE;

    output_counter_host[0] = 0;
    max_score_host[0] = 2;
    gpu_assert(hipMemcpyToSymbol(HIP_SYMBOL(device_memory), device_memory_host, 2 * sizeof(uint64_t)));
    gpu_assert(hipDeviceSynchronize())


    if (mode == 0 || mode == 1) {
        gpu_assert(hipMalloc(&block_offsets, GRID_SIZE * sizeof(CurvePoint)))
        gpu_assert(hipMalloc(&offsets, (uint64_t)GRID_SIZE * BLOCK_SIZE * sizeof(CurvePoint)))
        thread_offsets_host = new CurvePoint[BLOCK_SIZE];
        gpu_assert(hipHostAlloc(&thread_offsets_host, BLOCK_SIZE * sizeof(CurvePoint), hipHostMallocWriteCombined))
    }

    _uint256 max_key;
    if (mode == 0 || mode == 1) {
        _uint256 GRID_WORK = cpu_mul_256_mod_p(cpu_mul_256_mod_p(_uint256{0, 0, 0, 0, 0, 0, 0, THREAD_WORK}, _uint256{0, 0, 0, 0, 0, 0, 0, BLOCK_SIZE}), _uint256{0, 0, 0, 0, 0, 0, 0, GRID_SIZE});
        max_key = _uint256{0x7FFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF, 0x5D576E73, 0x57A4501D, 0xDFE92F46, 0x681B20A0};
        max_key = cpu_sub_256(max_key, GRID_WORK);
        max_key = cpu_sub_256(max_key, _uint256{0, 0, 0, 0, 0, 0, 0, THREAD_WORK});
        max_key = cpu_add_256(max_key, _uint256{0, 0, 0, 0, 0, 0, 0, 2});
    } else if (mode == 2) {
        max_key = _uint256{0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF};
    }

    _uint256 base_random_key{0, 0, 0, 0, 0, 0, 0, 0};
    _uint256 random_key_increment{0, 0, 0, 0, 0, 0, 0, 0};
    int status;
    if (mode == 0 || mode == 1) {
        status = generate_secure_random_key(base_random_key, max_key, 255);
        random_key_increment = cpu_mul_256_mod_p(cpu_mul_256_mod_p(uint32_to_uint256(BLOCK_SIZE), uint32_to_uint256(GRID_SIZE)), uint32_to_uint256(THREAD_WORK));
    } else if (mode == 2) {
        status = generate_secure_random_key(base_random_key, max_key, 256);
        random_key_increment = cpu_mul_256_mod_p(cpu_mul_256_mod_p(uint32_to_uint256(BLOCK_SIZE), uint32_to_uint256(GRID_SIZE)), uint32_to_uint256(THREAD_WORK));
        base_random_key.h &= ~(THREAD_WORK - 1);
    }

    if (status) {
        message_queue_mutex.lock();
        message_queue.push(Message{milliseconds(), 10 + status});
        message_queue_mutex.unlock();
        return;
    }
    _uint256 random_key = base_random_key;

    if (mode == 0 || mode == 1) {
        CurvePoint* addends_host = new CurvePoint[THREAD_WORK - 1];
        CurvePoint p = G;
        for (int i = 0; i < THREAD_WORK - 1; i++) {
            addends_host[i] = p;
            p = cpu_point_add(p, G);
        }
        gpu_assert(hipMemcpyToSymbol(HIP_SYMBOL(addends), addends_host, (THREAD_WORK - 1) * sizeof(CurvePoint)))
        delete[] addends_host;

        CurvePoint* block_offsets_host = new CurvePoint[GRID_SIZE];
        CurvePoint block_offset = cpu_point_multiply(G, _uint256{0, 0, 0, 0, 0, 0, 0, THREAD_WORK * BLOCK_SIZE});
        p = G;
        for (int i = 0; i < GRID_SIZE; i++) {
            block_offsets_host[i] = p;
            p = cpu_point_add(p, block_offset);
        }
        gpu_assert(hipMemcpy(block_offsets, block_offsets_host, GRID_SIZE * sizeof(CurvePoint), hipMemcpyHostToDevice))
        delete[] block_offsets_host;
    }

    if (mode == 0 || mode == 1) {
        hipStream_t streams[2];
        gpu_assert(hipStreamCreate(&streams[0]))
        gpu_assert(hipStreamCreate(&streams[1]))
        
        _uint256 previous_random_key = random_key;
        bool first_iteration = true;
        uint64_t start_time;
        uint64_t end_time;
        double elapsed;

        while (true) {
            if (!first_iteration) {
                if (mode == 0) {
                    gpu_address_work<<<GRID_SIZE, BLOCK_SIZE, 0, streams[0]>>>(score_method, offsets);
                } else {
                    gpu_contract_address_work<<<GRID_SIZE, BLOCK_SIZE, 0, streams[0]>>>(score_method, offsets);
                }
            }

            if (!first_iteration) {
                previous_random_key = random_key;
                random_key = cpu_add_256(random_key, random_key_increment);
                if (gte_256(random_key, max_key)) {
                    random_key = cpu_sub_256(random_key, max_key);
                }
            }
            CurvePoint thread_offset = cpu_point_multiply(G, _uint256{0, 0, 0, 0, 0, 0, 0, THREAD_WORK});
            CurvePoint p = cpu_point_multiply(G, cpu_add_256(_uint256{0, 0, 0, 0, 0, 0, 0, THREAD_WORK - 1}, random_key));
            for (int i = 0; i < BLOCK_SIZE; i++) {
                thread_offsets_host[i] = p;
                p = cpu_point_add(p, thread_offset);
            }
            gpu_assert(hipMemcpyToSymbolAsync(HIP_SYMBOL(thread_offsets), thread_offsets_host, BLOCK_SIZE * sizeof(CurvePoint), 0, hipMemcpyHostToDevice, streams[1]));
            gpu_assert(hipStreamSynchronize(streams[1]))
            gpu_assert(hipStreamSynchronize(streams[0]))

            if (!first_iteration) {
                end_time = milliseconds();
                elapsed = (end_time - start_time) / 1000.0;
            }
            start_time = milliseconds();

            gpu_address_init<<<GRID_SIZE/BLOCK_SIZE, BLOCK_SIZE, 0, streams[0]>>>(block_offsets, offsets);
            if (!first_iteration) {
                gpu_assert(hipMemcpyFromSymbolAsync(device_memory_host, HIP_SYMBOL(device_memory), (2 + OUTPUT_BUFFER_SIZE * 3) * sizeof(uint64_t), 0, hipMemcpyDeviceToHost, streams[1]))
                gpu_assert(hipStreamSynchronize(streams[1]))
            }
            if (!first_iteration) {
                global_max_score_mutex.lock();
                if (output_counter_host[0] != 0) {
                    if (max_score_host[0] > global_max_score) {
                        global_max_score = max_score_host[0];
                    } else {
                        max_score_host[0] = global_max_score;
                    }
                }
                global_max_score_mutex.unlock();

                double speed = GRID_WORK / elapsed / 1000000.0 * 2;
                if (output_counter_host[0] != 0) {
                    int valid_results = 0;

                    for (int i = 0; i < output_counter_host[0]; i++) {
                        if (output_buffer2_host[i] < max_score_host[0]) { continue; }
                        valid_results++;
                    }

                    if (valid_results > 0) {
                        _uint256* results = new _uint256[valid_results];
                        int* scores = new int[valid_results];
                        valid_results = 0;

                        for (int i = 0; i < output_counter_host[0]; i++) {
                            if (output_buffer2_host[i] < max_score_host[0]) { continue; }

                            uint64_t k_offset = output_buffer_host[i];
                            _uint256 k = cpu_add_256(previous_random_key, cpu_add_256(_uint256{0, 0, 0, 0, 0, 0, 0, THREAD_WORK}, _uint256{0, 0, 0, 0, 0, 0, (uint32_t)(k_offset >> 32), (uint32_t)(k_offset & 0xFFFFFFFF)}));

                            if (output_buffer3_host[i]) {
                                k = cpu_sub_256(N, k);
                            }
                
                            int idx = valid_results++;
                            results[idx] = k;
                            scores[idx] = output_buffer2_host[i];
                        }

                        message_queue_mutex.lock();
                        message_queue.push(Message{end_time, 0, device_index, hipSuccess, speed, valid_results, results, scores});
                        message_queue_mutex.unlock();
                    } else {
                        message_queue_mutex.lock();
                        message_queue.push(Message{end_time, 0, device_index, hipSuccess, speed, 0});
                        message_queue_mutex.unlock();
                    }
                } else {
                    message_queue_mutex.lock();
                    message_queue.push(Message{end_time, 0, device_index, hipSuccess, speed, 0});
                    message_queue_mutex.unlock();
                }
            }

            if (!first_iteration) {
                output_counter_host[0] = 0;
                gpu_assert(hipMemcpyToSymbolAsync(HIP_SYMBOL(device_memory), device_memory_host, sizeof(uint64_t), 0, hipMemcpyHostToDevice, streams[1]));
                gpu_assert(hipStreamSynchronize(streams[1]))
            }
            gpu_assert(hipStreamSynchronize(streams[0]))
            first_iteration = false;
        }
    }

    if (mode == 2) {
        while (true) {
            uint64_t start_time = milliseconds();
            gpu_contract2_address_work<<<GRID_SIZE, BLOCK_SIZE>>>(score_method, create2_factory, random_key, bytecode);

            gpu_assert(hipDeviceSynchronize())
            gpu_assert(hipMemcpyFromSymbol(device_memory_host, HIP_SYMBOL(device_memory), (2 + OUTPUT_BUFFER_SIZE * 3) * sizeof(uint64_t)))

            uint64_t end_time = milliseconds();
            double elapsed = (end_time - start_time) / 1000.0;

            global_max_score_mutex.lock();
            if (output_counter_host[0] != 0) {
                if (max_score_host[0] > global_max_score) {
                    global_max_score = max_score_host[0];
                } else {
                    max_score_host[0] = global_max_score;
                }
            }
            global_max_score_mutex.unlock();

            double speed = GRID_WORK / elapsed / 1000000.0;
            if (output_counter_host[0] != 0) {
                int valid_results = 0;

                for (int i = 0; i < output_counter_host[0]; i++) {
                    if (output_buffer2_host[i] < max_score_host[0]) { continue; }
                    valid_results++;
                }

                if (valid_results > 0) {
                    _uint256* results = new _uint256[valid_results];
                    int* scores = new int[valid_results];
                    valid_results = 0;

                    for (int i = 0; i < output_counter_host[0]; i++) {
                        if (output_buffer2_host[i] < max_score_host[0]) { continue; }

                        uint64_t k_offset = output_buffer_host[i];
                        _uint256 k = cpu_add_256(random_key, _uint256{0, 0, 0, 0, 0, 0, (uint32_t)(k_offset >> 32), (uint32_t)(k_offset & 0xFFFFFFFF)});
            
                        int idx = valid_results++;
                        results[idx] = k;
                        scores[idx] = output_buffer2_host[i];
                    }

                    message_queue_mutex.lock();
                    message_queue.push(Message{end_time, 0, device_index, hipSuccess, speed, valid_results, results, scores});
                    message_queue_mutex.unlock();
                } else {
                    message_queue_mutex.lock();
                    message_queue.push(Message{end_time, 0, device_index, hipSuccess, speed, 0});
                    message_queue_mutex.unlock();
                }
            } else {
                message_queue_mutex.lock();
                message_queue.push(Message{end_time, 0, device_index, hipSuccess, speed, 0});
                message_queue_mutex.unlock();
            }

            random_key = cpu_add_256(random_key, random_key_increment);

            output_counter_host[0] = 0;
            gpu_assert(hipMemcpyToSymbol(HIP_SYMBOL(device_memory), device_memory_host, sizeof(uint64_t)));
        }
    }
}


void print_speeds(int num_devices, int* device_ids, double* speeds) {
    double total = 0.0;
    for (int i = 0; i < num_devices; i++) {
        total += speeds[i];
    }

    printf("Total: %.2fM/s", total);
    for (int i = 0; i < num_devices; i++) {
        printf("    Device %d: %.2fM/s", device_ids[i], speeds[i]);
    }
}


int main(int argc, char *argv[]) {
    int num_devices = 0;
    int device_ids[10];

    int score_method = -1; // 0 = leading zeroes, 1 = zeros

    int mode = -1; // 0 = EOA, 1 = CREATE, 2 = CREATE2
    char* input_create2_factory = 0;
    char* input_bytecode_path = 0;
    bool usingC0ntractcharm = false;

    for (int i = 1; i < argc;) {
        if (strcmp(argv[i], "--device") == 0 || strcmp(argv[i], "-d") == 0) {
            device_ids[num_devices++] = atoi(argv[i + 1]);
            i += 2;
        } else if (strcmp(argv[i], "--work-scale") == 0 || strcmp(argv[i], "-w") == 0) {
            GRID_SIZE = 1U << atoi(argv[i + 1]);
            i += 2;
        } else if (strcmp(argv[i], "--leading-zeros") == 0 || strcmp(argv[i], "-lz") == 0) {
            score_method = 0;
            i++;
        } else if (strcmp(argv[i], "--zeros") == 0 || strcmp(argv[i], "-z") == 0) {
            score_method = 1;
            i++;
        } else if (strcmp(argv[i], "--eoa") == 0 || strcmp(argv[i], "-e") == 0) {
            mode = 0;
            i++;
        } else if (strcmp(argv[i], "--create") == 0 || strcmp(argv[i], "-c") == 0) {
            mode = 1;
            i++;
        } else if (strcmp(argv[i], "--create2") == 0 || strcmp(argv[i], "-c2") == 0) {
            mode = 2;
            input_create2_factory = argv[i + 1];
            input_bytecode_path = argv[i + 2];
            if (!input_create2_factory) {
                printf("You must specify a CREATE2 factory when using -c2(\"-c2 <factory> <bytecode_path>\")\n");
                return 1;
            }
            if (strlen(input_create2_factory) != 40 && strlen(input_create2_factory) != 42) {
                printf("The CREATE2 factory must be 40 characters long\n");
                return 1;
            }
            if (!input_bytecode_path) {
                printf("You must specify contract bytecode path when using -c2(\"-c2 <factory> <bytecode_path>\")\n");
                return 1;
            }
            i += 3;
        } else if (strcmp(argv[i], "--c0ntractcharm") == 0 || strcmp(argv[i], "-cc") == 0) {
            mode = 2;
            input_create2_factory = argv[i + 1];
            if (!input_create2_factory) {
                printf("You must specify the C0ntractcharm address when using -cc(\"-cc <address>\")\n");
                return 1;
            }
            if (strlen(input_create2_factory) != 40 && strlen(input_create2_factory) != 42) {
                printf("The C0ntractcharm address must be 40 characters long\n");
                return 1;
            }
            usingC0ntractcharm = true;
            i += 2;
        } else {
            i++;
        }
    }

    if (num_devices == 0) {
        printf("No device is specified(\"-d 0\" to use device 0)\n");
        return 1;
    }
    for (int i = 0; i < num_devices; i++) {
        hipError_t e = hipSetDevice(device_ids[i]);
        if (e != hipSuccess) {
            printf("Could not detect device: %d\n", device_ids[i]);
            return 1;
        }
    }

    if (score_method == -1) {
        printf("No scoring method is specified(\"-lz\" for leading zeros, \"-z\" for zeros)\n");
        return 1;
    }

    if (mode == -1) {
        printf("No mode is specified(\"-e\" for EOA, \"-c\" for CREATE Contract, \"-c2\" for CREATE2 Contract)\", \"-cc <address>\" for C0ntractcharm)\n");
        return 1;
    }
    if (mode != 0 && mode != 1 && mode != 2) {
        printf("Mode is invalid: %d\n", mode);
        return 1;
    }

    #define nothex(n) ((n < 48 || n > 57) && (n < 65 || n > 70) && (n < 97 || n > 102))
    _uint256 bytecode_hash;
    if (mode == 2) {
        if (usingC0ntractcharm) {
            //0x6000600060006000335afa3d600060003e3d6000f3 21 bytes
            uint8_t* bytecode = new uint8_t[21];
            if (bytecode == 0) {
                printf("Failed to allocate memory");
                return 1;
            }
            bytecode[0] = 0x60;
            bytecode[1] = 0x00;
            bytecode[2] = 0x60;
            bytecode[3] = 0x00;
            bytecode[4] = 0x60;
            bytecode[5] = 0x00;
            bytecode[6] = 0x60;
            bytecode[7] = 0x00;
            bytecode[8] = 0x33;
            bytecode[9] = 0x5a;
            bytecode[10] = 0xfa;
            bytecode[11] = 0x3d;
            bytecode[12] = 0x60;
            bytecode[13] = 0x00;
            bytecode[14] = 0x60;
            bytecode[15] = 0x00;
            bytecode[16] = 0x3e;
            bytecode[17] = 0x3d;
            bytecode[18] = 0x60;
            bytecode[19] = 0x00;
            bytecode[20] = 0xf3;

            bytecode_hash = cpu_full_keccak(bytecode, 21);
            delete[] bytecode;
        } else {
            std::ifstream infile(input_bytecode_path, std::ios::binary);
            if (!infile.is_open()) {
                printf("Failed to read the bytecode file\n");
                return 1;
            }

            int file_size = 0;
            {
                infile.seekg(0, std::ios::end);
                std::streampos file_size_ = infile.tellg();
                infile.seekg(0, std::ios::beg);
                file_size = file_size_ - infile.tellg();
            }

            if (file_size & 1) {
                printf("The length of bytecode in file is invalid\n");
                return 1;
            }

            uint8_t* bytecode = new uint8_t[24576];
            if (bytecode == 0) {
                printf("Failed to allocate memory");
                return 1;
            }

            char byte[2];
            bool prefix = false;
            for (int i = 0; i < (file_size >> 1); i++) {
                infile.read((char*)&byte, 2);
                if (i == 0) {
                    prefix = byte[0] == '0' && byte[1] == 'x';
                    if ((file_size >> 1) > (prefix ? 24577 : 24576)) {
                        printf("The bytecode in file is invalid\n");
                        delete[] bytecode;
                        return 1;
                    }
                    if (prefix) { continue; }
                }

                if (nothex(byte[0]) || nothex(byte[1])) {
                    printf("The bytecode in file is invalid\n");
                    delete[] bytecode;
                    return 1;
                }

                bytecode[i - prefix] = (uint8_t)strtol(byte, 0, 16);
            }
            bytecode_hash = cpu_full_keccak(bytecode, (file_size >> 1) - prefix);
            delete[] bytecode;
        }
    }

    Address create2_factory;
    if (mode == 2) {
        if (strlen(input_create2_factory) == 42) {
            input_create2_factory += 2;
        }
        char substr[9];

        #define round(i, offset) \
        strncpy(substr, input_create2_factory + offset * 8, 8); \
        if (nothex(substr[0]) || nothex(substr[1]) || nothex(substr[2]) || nothex(substr[3]) || nothex(substr[4]) || nothex(substr[5]) || nothex(substr[6]) || nothex(substr[7])) { \
            printf("The CREATE2 factory is invalid\n"); \
            return 1; \
        } \
        create2_factory.i = strtoull(substr, 0, 16);

        round(a, 0)
        round(b, 1)
        round(c, 2)
        round(d, 3)
        round(e, 4)

        #undef round
    }
    #undef nothex

    std::vector<std::thread> threads;
    uint64_t global_start_time = milliseconds();
    for (int i = 0; i < num_devices; i++) {
        std::thread th(host_thread, device_ids[i], i, score_method, mode, create2_factory, bytecode_hash);
        threads.push_back(move(th));
    }

    double speeds[100];
    while(true) {
        message_queue_mutex.lock();
        if (message_queue.size() == 0) {
            message_queue_mutex.unlock();
            std::this_thread::sleep_for(std::chrono::milliseconds(500));
        } else {
            while (!message_queue.empty()) {
                Message m = message_queue.front();
                message_queue.pop();

                int device_index = m.device_index;

                if (m.status == 0) {
                    speeds[device_index] = m.speed;

                    printf("\r");
                    if (m.results_count != 0) {
                        Address* addresses = new Address[m.results_count];
                        for (int i = 0; i < m.results_count; i++) {
                            if (mode == 0) {
                                CurvePoint p = cpu_point_multiply(G, m.results[i]);
                                addresses[i] = cpu_calculate_address(p.x, p.y);
                            } else if (mode == 1) {
                                CurvePoint p = cpu_point_multiply(G, m.results[i]);
                                addresses[i] = cpu_calculate_contract_address(cpu_calculate_address(p.x, p.y));
                            } else if (mode == 2) {
                                addresses[i] = cpu_calculate_contract_address2(create2_factory, m.results[i], bytecode_hash);
                            }
                        }

                        for (int i = 0; i < m.results_count; i++) {
                            _uint256 k = m.results[i];
                            int score = m.scores[i];
                            Address a = addresses[i];
                            uint64_t time = (m.time - global_start_time) / 1000;

                            if (mode == 0 || mode == 1) {
                                printf("Elapsed: %06u    Score: %02u    Private Key: 0x%08x%08x%08x%08x%08x%08x%08x%08x    Address: 0x%08x%08x%08x%08x%08x\n", (uint32_t)time, score, k.a, k.b, k.c, k.d, k.e, k.f, k.g, k.h, a.a, a.b, a.c, a.d, a.e);
                            } else if (mode == 2) {
                                printf("Elapsed: %06u    Score: %02u    Salt: 0x%08x%08x%08x%08x%08x%08x%08x%08x    Address: 0x%08x%08x%08x%08x%08x\n", (uint32_t)time, score, k.a, k.b, k.c, k.d, k.e, k.f, k.g, k.h, a.a, a.b, a.c, a.d, a.e);
                            }
                        }

                        delete[] addresses;
                        delete[] m.results;
                        delete[] m.scores;
                    }
                    print_speeds(num_devices, device_ids, speeds);
                    fflush(stdout);
                } else if (m.status == 1) {
                    printf("\rCuda error %d on device %d. Device will halt work.\n", m.error, device_ids[device_index]);
                    print_speeds(num_devices, device_ids, speeds);
                    fflush(stdout);
                } else if (m.status == 11) {
                    printf("\rError from BCryptGenRandom. Device %d will halt work.", device_ids[device_index]);
                    print_speeds(num_devices, device_ids, speeds);
                    fflush(stdout);
                } else if (m.status == 12) {
                    printf("\rError while reading from /dev/urandom. Device %d will halt work.", device_ids[device_index]);
                    print_speeds(num_devices, device_ids, speeds);
                    fflush(stdout);
                } else if (m.status == 13) {
                    printf("\rError while opening /dev/urandom. Device %d will halt work.", device_ids[device_index]);
                    print_speeds(num_devices, device_ids, speeds);
                    fflush(stdout);
                } else if (m.status == 100) {
                    printf("\Failed to allocate memory. Device %d will halt work.", device_ids[device_index]);
                }
                // break;
            }
            message_queue_mutex.unlock();
        }
    }
}